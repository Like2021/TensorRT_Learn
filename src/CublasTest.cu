#include "hipblas.h"
#include <hip/hip_runtime.h>

#define M 512
#define N 512
#define K 512

#define RUNTIME_CUDA_ERROR(errorInfo) CHECK_CUDA(errorInfo, hipSuccess)

#define CUDA_FREE(d_ptr){\
    if (d_ptr != nullptr) RUNTIME_CUDA_ERROR(hipFree(d_ptr)); d_ptr = nullptr;\
}\

int main(int argc, char* argv[])
{
    // 组织数据
    int A = M * N;


    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasDestroy(handle);

    return 0;
}